#include "hip/hip_runtime.h"
#include <hip/hip_complex.h>
#include "cutil.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>   // M_PI
#include "qft_gpu_v5.h"

#define BLOCKS 32768


// Menggunakan shared memory untuk target bawah (0 hingga 8)


__device__ static void qft_gpu_v5_single_lower_state(int tgt, unsigned long i, int width, unsigned long block_base, hipDoubleComplex *sh_v){
	unsigned long tgt_bit = (1ul << tgt);
	unsigned long i_other = i^tgt_bit;

	// Separuh warp pertama mendapat koefisien dari shared memory.
	hipDoubleComplex v_i, v_iother;
	if ((i % 16) >= 8) {
		v_i = sh_v[i - block_base];
		v_iother = sh_v[i_other - block_base];
	}

	// Hitung fase.
	unsigned long phase_coef = 1ul;
	unsigned long normal = (1ul << (width - tgt - 1));

	unsigned long ctl_bit = (1ul << tgt);
	for (int ctl = tgt + 1; ctl < width; ctl++) {
		ctl_bit = (ctl_bit << 1);
		phase_coef = (phase_coef <<1);
		if ((i & ctl_bit) != 0) {
			phase_coef = phase_coef ^ 1ul;
		}
	}
	phase_coef = phase_coef ^ (normal);

	// Separuh warp kedua mendapat koefisien dari shared memory.
	if ((i % 16) < 8) {
		v_iother = sh_v[i_other-block_base];
		v_i = sh_v[i-block_base];
	}

	// Selesai menghitung gerbang.
	float phi = float(phase_coef) * float(M_PI) / float(normal);
	float c = cosf(phi);
	float s = sqrtf(1.0f-c*c);
	if (phi>float(M_PI)) {
		s *= -1.0f;
	}
	hipDoubleComplex phase = {c, s};
	v_i = hipCmul(v_i, phase);

	hipDoubleComplex ai, aother;	// koefisien i dan (i^tgt_bit)

	hipDoubleComplex cuM_SQRT1_2 = make_hipDoubleComplex(M_SQRT1_2, 0);
	ai = hipCmul(cuM_SQRT1_2, hipCsub(v_iother, v_i));
	aother = hipCmul(cuM_SQRT1_2, hipCadd(v_iother, v_i));


	if ((i % 16) >= 8) {
		sh_v[i - block_base] = ai;
		sh_v[i_other - block_base] = aother;
	} else {
		sh_v[i_other - block_base] = aother;
		sh_v[i - block_base] = ai;
	}
}


// Mengaplikasikan geser fase dan transformasi Hadamard untuk qubit 'tgt' dan state 'i'.
// Catatan: HANYA dipanggil dengan ((i & tgt_bit) == 1)
__device__ static void qft_gpu_v5_single_state(int tgt, unsigned long i, int width, hipDoubleComplex *v){
	unsigned long phase_coef = 1ul;
	unsigned long tgt_bit = (1ul << tgt);
	unsigned long i_other = i^tgt_bit;
	unsigned long normal = (1ul << (width - tgt - 1));

/*
	if ((i & tgt_bit) == 0) {
		return;
	}
*/

	// Catatan: Geser fase (dengan target sama) commute.
	unsigned long ctl_bit = (1ul << tgt);
	for (int ctl=tgt+1; ctl<width; ctl++) {
		ctl_bit = (ctl_bit << 1);
		phase_coef = (phase_coef <<1);
		if ((i & ctl_bit) != 0) {
			phase_coef = phase_coef ^ 1ul;
		}
	}
	phase_coef = phase_coef ^ (normal);
	float phi = float(phase_coef) * float(M_PI) / float(normal);
	float c = cosf(phi);
	float s = sqrtf(1.0f - c * c);
	if (phi>float(M_PI)) {
		s *= -1.0f;
	}
	hipDoubleComplex phase = {c, s};
	hipDoubleComplex v_i = v[i];
	hipDoubleComplex v_iother = v[i_other];
	v_i = hipCmul(v_i, phase);

	hipDoubleComplex ai, aother;	// koefisien i dan (i^tgt_bit)

	hipDoubleComplex cuM_SQRT1_2 = make_hipDoubleComplex(M_SQRT1_2, 0);
	ai = hipCmul(cuM_SQRT1_2, hipCsub(v_iother, v_i));
	aother = hipCmul(cuM_SQRT1_2, hipCadd(v_iother, v_i));

	v[i] = ai;
	v[i_other] = aother;
}


// Kernel ini melakukan QFT single stage.
__global__ static void K_qft_gpu_v5_stage(int width, hipDoubleComplex *v, int tgt){
	unsigned long N = (1ul << width);

	// Membagi threads ke tiap state.
	unsigned long long bidx = blockIdx.y*gridDim.x + blockIdx.x;
	unsigned long long i = bidx*blockDim.x + threadIdx.x;

	if (i >= N) {
		return;
	}

	unsigned long tgt_bit = (1ul << tgt);
	if ((i & tgt_bit) != 0) {
		qft_gpu_v5_single_state(tgt, i, width, v);
	}
}




// Kernel ini melakukan QFT single stage.
__global__ static void K_qft_gpu_v5_stage_0to8(int width, hipDoubleComplex *v){
	unsigned long N = (1ul << width);

	// Membagi threads ke tiap state.
	unsigned long long bidx = blockIdx.y*gridDim.x + blockIdx.x;
	unsigned long long i = bidx*blockDim.x + threadIdx.x;

	// salin ke dalam
	unsigned long block_base = bidx*blockDim.x;
	__shared__ hipDoubleComplex sh_v[512]; // Note: hardcoded shared memory size
	if (i < N) {
		sh_v[threadIdx.x] = v[i];
	}
	__syncthreads();


	// hitung
	for (int tgt = min(width-1,8); tgt >= 0; tgt--) {
		unsigned long tgt_bit = (1ul << tgt);
		if ( (i < N) && ((i & tgt_bit) != 0) ) {
			qft_gpu_v5_single_lower_state(tgt, i, width, block_base, sh_v);
		}
		__syncthreads();
	}

	// salin ke luar
	if (i < N) {
		v[i] = sh_v[threadIdx.x];
	}
}


// Implementasi QFT gerbang demi gerbang menggunakan GPU.
void qft_gpu_v5_helper(int width, hipDoubleComplex *d_v, int threadsPerBlock){
	unsigned long N = (1ul << width);

	unsigned long long nblocks = (N+threadsPerBlock-1)/threadsPerBlock;
	unsigned long long xblocks, yblocks;

	yblocks = (nblocks+BLOCKS-1)/BLOCKS;
	xblocks = BLOCKS;
	if (nblocks < xblocks) {
		xblocks = nblocks;
	}
	dim3 blocks(xblocks, yblocks);

	// Utk tiap qubit...
	int tgt;
	for (tgt=width-1; tgt>=9; tgt--) {
		K_qft_gpu_v5_stage<<<blocks, threadsPerBlock>>>(width, d_v, tgt);
		CUT_CHECK_ERROR("K_qft_gpu_v5_stage gagal.");
	}
	K_qft_gpu_v5_stage_0to8<<<blocks, threadsPerBlock>>>(width, d_v);
	CUT_CHECK_ERROR("K_qft_gpu_v5_stage_0to8 gagal.");
}
