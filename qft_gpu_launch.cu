#include "hip/hip_runtime.h"
/*Pustaka CUDA*/
#include <hip/hip_complex.h>

/*Pustaka standar C/C++*/
#include <stdio.h>
#include <stdlib.h>

/*Para implementasi qft*/
#include "qft_gpu_launch.h"
#include "qft_gpu_v1.h"
#include "qft_gpu_v2.h"
#include "qft_gpu_v3.h"
#include "qft_gpu_v4.h"
#include "qft_gpu_v5.h"
#include "qft_gpu_v6.h"
#include "qft_host.h"

/*Utilitas-utilitas*/
#include "Stopwatch.h"
#include "quantum_utils.h"
#include "cutil.h"



/*
 * Menyalin 'h_v' ke dalam memori device, menjalankan qft_gpu_bygate_v1_helper() di situ,
 * lalu menyalin balik hasilnya, serta me-return CUDA time yang digunakan, dalam detik.
 */

double qft_gpu(int width, hipDoubleComplex *h_v, int threadsPerBlock, QFT_versions vers)
{
	int deviceCount;
	CUDA_SAFE_CALL(hipGetDeviceCount(&deviceCount));

	if (deviceCount == 0) {
		fprintf(stderr, "Tidak ditemukan perangkat yang mendukung CUDA\n");
		exit(1);
	}
	hipDeviceProp_t deviceProp;
	CUDA_SAFE_CALL(hipGetDeviceProperties(&deviceProp, 0));

	hipSetDevice(0);

	threadsPerBlock = min(threadsPerBlock, deviceProp.maxThreadsPerBlock);

	unsigned long long N = (1ull << width);

	hipDoubleComplex *d_v = NULL;

	if (vers != QFT_v0_HOST) {
		if (hipMalloc((void**)&d_v, sizeof(hipDoubleComplex)*N) != hipSuccess) {
			fprintf(stderr, "Error mengalokasikan %llu keadaan dalam memori perangkat.\n", N);
			exit(1);
		}
	}

	//unsigned int htimer;//obsolete/usang
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	//cutCreateTimer(&htimer);//obsolete/usang

	if (vers != QFT_v0_HOST) {
		hipMemcpy((void*)d_v, (void*)h_v, sizeof(hipDoubleComplex)*N, hipMemcpyHostToDevice);
		hipDeviceSynchronize();
	}


	// Jalankan QFT.
	printf("Menjalankan GPU-QFT v%d dengan %d qubit\n", (int)vers, width);
	fflush(stdout);

	Stopwatch s(false); // CPU time
	//cutResetTimer(htimer); //obsolete/usang
	//cutStartTimer(htimer); //obsolete/usang
	hipEventRecord(start, 0);
	switch(vers) {
	case QFT_v0_HOST:
		//TODO: count CPU time
		s.restart();
		qft_host(width, h_v);
		s.stop();
		break;

	case QFT_v1_PLAIN:
		qft_gpu_v1_helper(width, d_v, threadsPerBlock);
		break;

	case QFT_v2_PHASE:
		qft_gpu_v2_helper(width, d_v, threadsPerBlock);
		break;

	case QFT_v3_TRIG:
		qft_gpu_v3_helper(width, d_v, threadsPerBlock);
		break;

	case QFT_v4_SHARED:
		qft_gpu_v4_helper(width, d_v, threadsPerBlock);
		break;

	case QFT_v5_0TO8:
		qft_gpu_v5_helper(width, d_v, threadsPerBlock);
		break;

	case QFT_v6_GROUPED:
		qft_gpu_v6_helper(width, d_v, threadsPerBlock);
		break;

	default:
		fprintf(stderr, "Versi implementasi GPU QFT yang diminta, %d, tidak dikenali.\n", (int)vers);
		exit(1);
	}

	//hipDeviceSynchronize(); //obsolete/usang
	//cutStopTimer(htimer); //obsolete/usang
	hipEventRecord(stop, 0);

	if (vers != QFT_v0_HOST) {
		hipMemcpy((void*)h_v, (void*)d_v, sizeof(hipDoubleComplex)*N, hipMemcpyDeviceToHost);
	hipEventSynchronize(stop);
		hipDeviceSynchronize();
		hipFree(d_v);
	}

	float runtime_ms;
	if (vers == QFT_v0_HOST) {
		runtime_ms = s.getElapsed()*1000.0;
	} else {
		//runtime_ms = cutGetTimerValue(htimer); //obsolete/usang
	hipEventElapsedTime(&runtime_ms, start, stop);
	}
	hipEventDestroy(start);
	hipEventDestroy(stop);

	return double(runtime_ms/1000.0);
}


double qft_gpu_qureg(quantum_reg& qr, int threadsPerBlock, QFT_versions vers)
{
	// Ekspansi qr.
	hipDoubleComplex *v = NULL;
	qutil_expand_qr(qr, &v);

	double runtime_ms = qft_gpu(qr.width, v, threadsPerBlock, vers);

	// Hapus qr yang masih ada, diganti dengan mengekspansi vektor yang dikalkulasi
	quantum_delete_qureg(&qr);
	qr = qutil_collapse_qr(qr.width, v);

	// Bersih-bersih memori
	qutil_destroy_qvec(&v);
	return runtime_ms;
}
