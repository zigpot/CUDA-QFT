#include "hip/hip_runtime.h"
#include <hip/hip_complex.h>
#include "cutil.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>   // M_PI
#include "qft_gpu_v4.h"

#define BLOCKS 32768


// Like v3, but by combining the phase shift and hadamard gates, we can remove
// one read and write to/from global memory.


// This applies the phase shifts and Hadamard transform for qubit 'tgt' and state 'i'.
// Note: This should ONLY be called with ((i & tgt_bit) == 1)
__device__ static void qft_gpu_v4_single_state(int tgt, unsigned long i, int width,  hipDoubleComplex *v){
	unsigned long phase_coef = 1ul;
	unsigned long tgt_bit = (1ul << tgt);
	unsigned long i_other = i^tgt_bit;
	unsigned long normal = (1ul << (width - tgt - 1));

	if ((i & tgt_bit) == 0) {
        // This function should not have been called in this case.
		return;
	}

	unsigned long ctl_bit = (1ul << tgt);
	for (int ctl = tgt + 1; ctl < width; ctl++) {
		ctl_bit = (ctl_bit << 1);
		phase_coef = (phase_coef <<1);
		if ((i & ctl_bit) != 0) {
			phase_coef = phase_coef ^ 1ul;
		}
	}
	phase_coef = phase_coef ^ (normal);
	float phi = float(phase_coef) * float(M_PI) / float(normal);

	float c = cosf(phi);
	float s = sqrtf(1.0f - c * c);
	if (phi>float(M_PI)) {
		s *= -1.0f;
	}
	hipDoubleComplex phase = {c, s};

	hipDoubleComplex v_i = v[i];
	hipDoubleComplex v_iother = v[i_other];
	v_i = hipCmul(v_i, phase);

	hipDoubleComplex ai, aother;    // coefficients i and (i^tgt_bit)

	hipDoubleComplex cuM_SQRT1_2 = make_hipDoubleComplex(M_SQRT1_2, 0);
	ai = hipCmul(cuM_SQRT1_2, hipCsub(v_iother, v_i));
	aother = hipCmul(cuM_SQRT1_2, hipCadd(v_iother, v_i));

	v[i] = ai;
	v[i_other] = aother;
}


// This kernel performs a single stage of the QFT.
__global__ static void K_qft_gpu_v4_stage(int width, hipDoubleComplex *v, int tgt){
	unsigned long N = (1ul << width);

    // Split threads over states.
	unsigned long long bidx = blockIdx.y*gridDim.x + blockIdx.x;
	unsigned long long i = bidx*blockDim.x + threadIdx.x;

	if (i >= N) {
		return;
	}

	unsigned long tgt_bit = (1ul << tgt);
	if ((i & tgt_bit) != 0) {
		qft_gpu_v4_single_state(tgt, i, width, v);
	}

}


// Implement the QFT gate by gate using the GPU.
void qft_gpu_v4_helper(int width, hipDoubleComplex *d_v, int threadsPerBlock){
	unsigned long N = (1ul << width);

	unsigned long long nblocks = (N+threadsPerBlock-1)/threadsPerBlock;
	unsigned long long xblocks, yblocks;

	yblocks = (nblocks + BLOCKS - 1)/BLOCKS;
	xblocks = BLOCKS;
	if (nblocks < xblocks) {
		xblocks = nblocks;
	}
	dim3 blocks(xblocks, yblocks);

    // For each qubit...
	int tgt;
	for (tgt = width - 1; tgt >= 0; tgt--) {
		K_qft_gpu_v4_stage<<<blocks, threadsPerBlock>>>(width, d_v, tgt);
		CUT_CHECK_ERROR("K_qft_gpu_v4_stage failed.");
	}
}
