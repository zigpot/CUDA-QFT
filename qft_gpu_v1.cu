#include "hip/hip_runtime.h"
#include <hip/hip_complex.h>
#include "cutil.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>   // M_PI
#include "qft_gpu_v1.h"

#define BLOCKSIZE 65535


// This is the original, straightforward port from CPU to GPU code with no optimizations.

// process bits i and i^(1ul << target)
// Note: This should ONLY be called with ((i & tgt_bit) == 1)
__device__ static void phase_shift_gpu(int ctl, int tgt, unsigned long i, int width, hipDoubleComplex *v)
{
	unsigned long tgt_bit = (1ul << tgt);
	if ((i & tgt_bit) == 0) {
        	// This function should not have been called in this case.
		return;
	}

	unsigned long ctl_bit = (1ul << ctl);

	float phi = 1.0f * float(M_PI) / float(1ul << (ctl - tgt));
	hipDoubleComplex phase = { cosf(phi), sinf(phi) };

	if ((i & ctl_bit) != 0) {
		v[i] = hipCmul(v[i], phase);
		// v[i^tgt_bit] stays unchanged
	}
}


// process bits i and i^(1ul << tgt)
// Note: This should ONLY be called with ((i & tgt_bit) == 1)
__device__ static void hadamard_gpu(unsigned long tgt, unsigned long i, int width, hipDoubleComplex *v)
{
	unsigned long tgt_bit = (1ul << tgt);
	if ((i & tgt_bit) == 0) {
		// This function should not have been called in this case.
		return;
	}

	unsigned long i_other = i^tgt_bit;
	hipDoubleComplex ai, aother;	// coefficient i and (i^tgt_bit)
	hipDoubleComplex v_i = v[i];
	hipDoubleComplex v_iother = v[i_other];

	hipDoubleComplex cuM_SQRT1_2 = make_hipDoubleComplex(M_SQRT1_2, 0);
	ai = hipCmul(cuM_SQRT1_2, hipCsub(v_iother, v_i));
	aother = hipCmul(cuM_SQRT1_2, hipCadd(v_iother, v_i));

	v[i] = ai;
	v[i_other] = aother;
}


// This applies the phase shifts and Hadamard transform for qubit 'tgt' and state 'i'.
// Note: This should ONLY be called with ((i & tgt_bit) == 1)
__device__ static void qft_gpu_v1_single_state(int tgt, unsigned long i, int width, hipDoubleComplex *v)
{
	unsigned long tgt_bit = (1ul << tgt);
	if ((i & tgt_bit) == 0) {
		return;
	}

    // Note: Phase shifts (with the same target) commute.
	for (int ctl = tgt + 1; ctl < width; ctl++) {
		phase_shift_gpu(ctl, tgt, i, width, v);
	}
	hadamard_gpu(tgt, i, width, v);
}


// This kernel performs a single stage of the QFT.
__global__ static void K_qft_gpu_v1_stage(int width, hipDoubleComplex *v, int tgt)
{
	unsigned long N = (1ul << width);

    // Split threads over states.
	unsigned long long bidx = blockIdx.y*gridDim.x + blockIdx.x;
	unsigned long long i = bidx*blockDim.x + threadIdx.x;

	if (i >= N) {
		return;
	}

	unsigned long tgt_bit = (1ul << tgt);
	if ((i & tgt_bit) != 0) {
		qft_gpu_v1_single_state(tgt, i, width, v);
	}
}

// Implement the QFT gate by gate using the GPU.
void qft_gpu_v1_helper(int width, hipDoubleComplex *d_v, int threadsPerBlock)
{
	unsigned long N = (1ul << width);

	unsigned long long nblocks = (N+threadsPerBlock-1)/threadsPerBlock;
	unsigned long long xblocks, yblocks;
	yblocks = (nblocks+BLOCKSIZE-1)/BLOCKSIZE;
	xblocks = BLOCKSIZE;
	if (nblocks < xblocks) {
		xblocks = nblocks;
	}
	dim3 blocks(xblocks, yblocks);

    // For each qubit...
	for (int tgt = width - 1; tgt >= 0; tgt--) {
		K_qft_gpu_v1_stage<<<blocks, threadsPerBlock>>>(width, d_v, tgt);
		CUT_CHECK_ERROR("K_qft_gpu_v1_stage failed.");
	}
}
