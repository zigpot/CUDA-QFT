#include "hip/hip_runtime.h"
#include <hip/hip_complex.h>
#include "cutil.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>   // M_PI
#include "qft_gpu_v1.h"

#define BLOCKSIZE 65535


// Ini versi asli, di-port dari kode CPU ke GPU tanpa optimisasi.

// memproses bit i dan i^(1ul << target)
// Catatan: HANYA dipanggil dengan ((i & tgt_bit) == 1)
__device__ static void phase_shift_gpu(int ctl, int tgt, unsigned long i, int width, hipDoubleComplex *v)
{
	unsigned long tgt_bit = (1ul << tgt);
	if ((i & tgt_bit) == 0) {
		return;
	}

	unsigned long ctl_bit = (1ul << ctl);

	float phi = 1.0f * float(M_PI) / float(1ul << (ctl - tgt));
	hipDoubleComplex phase = { cosf(phi), sinf(phi) };

	if ((i & ctl_bit) != 0) {
		v[i] = hipCmul(v[i], phase);
		// v[i^tgt_bit] stays unchanged
	}
}


// memproses bit i dan i^(1ul << target)
// Catatan: HANYA dipanggil dengan ((i & tgt_bit) == 1)
__device__ static void hadamard_gpu(unsigned long tgt, unsigned long i, int width, hipDoubleComplex *v)
{
	unsigned long tgt_bit = (1ul << tgt);
	if ((i & tgt_bit) == 0) {
		return;
	}

	unsigned long i_other = i^tgt_bit;
	hipDoubleComplex ai, aother;	// koefisien i dan (i^tgt_bit)
	hipDoubleComplex v_i = v[i];
	hipDoubleComplex v_iother = v[i_other];

	hipDoubleComplex cuM_SQRT1_2 = make_hipDoubleComplex(M_SQRT1_2, 0);
	ai = hipCmul(cuM_SQRT1_2, hipCsub(v_iother, v_i));
	aother = hipCmul(cuM_SQRT1_2, hipCadd(v_iother, v_i));

	v[i] = ai;
	v[i_other] = aother;
}


// Mengaplikasikan geser fase dan transformasi Hadamard untuk qubit 'tgt' dan state 'i'.
// Catatan: HANYA dipanggil dengan ((i & tgt_bit) == 1)
__device__ static void qft_gpu_v1_single_state(int tgt, unsigned long i, int width, hipDoubleComplex *v)
{
	unsigned long tgt_bit = (1ul << tgt);
	if ((i & tgt_bit) == 0) {
		return;
	}

	// Catatan: Geser fase (dengan target sama) commute.
	for (int ctl = tgt + 1; ctl < width; ctl++) {
		phase_shift_gpu(ctl, tgt, i, width, v);
	}
	hadamard_gpu(tgt, i, width, v);
}


// Kernel ini melakukan QFT single stage.
__global__ static void K_qft_gpu_v1_stage(int width, hipDoubleComplex *v, int tgt)
{
	unsigned long N = (1ul << width);

	// Membagi threads ke tiap state.
	unsigned long long bidx = blockIdx.y*gridDim.x + blockIdx.x;
	unsigned long long i = bidx*blockDim.x + threadIdx.x;

	if (i >= N) {
		return;
	}

	unsigned long tgt_bit = (1ul << tgt);
	if ((i & tgt_bit) != 0) {
		qft_gpu_v1_single_state(tgt, i, width, v);
	}
}


// Implementasi gerbang QFT dengan menggunakan GPU.
void qft_gpu_v1_helper(int width, hipDoubleComplex *d_v, int threadsPerBlock)
{
	unsigned long N = (1ul << width);

	unsigned long long nblocks = (N+threadsPerBlock-1)/threadsPerBlock;
	unsigned long long xblocks, yblocks;
	yblocks = (nblocks+BLOCKSIZE-1)/BLOCKSIZE;
	xblocks = BLOCKSIZE;
	if (nblocks < xblocks) {
		xblocks = nblocks;
	}
	dim3 blocks(xblocks, yblocks);

	// Untuk tiap qubit...
	for (int tgt = width - 1; tgt >= 0; tgt--) {
		K_qft_gpu_v1_stage<<<blocks, threadsPerBlock>>>(width, d_v, tgt);
		CUT_CHECK_ERROR("K_qft_gpu_v1_stage gagal.");
	}
}
