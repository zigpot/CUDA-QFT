#include "hip/hip_runtime.h"
#include <hip/hip_complex.h>
#include "cutil.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>   // M_PI
#include "qft_gpu_v6.h"

#define BLOCKS 32768


// Menggunakan shared memory untuk target bawah


__device__ static void qft_gpu_v6_single_lower_state(int tgt, unsigned long i, int width, unsigned long block_base, hipDoubleComplex *sh_v){
	unsigned long tgt_bit = (1ul << tgt);
	unsigned long i_other = i^tgt_bit;

	// Separuh warp pertama mendapat koefisien dari shared memory.
	hipDoubleComplex v_i, v_iother;
	if ((i%16)>=8) {
		v_i = sh_v[i-block_base];
		v_iother = sh_v[i_other-block_base];
	}

	// Hitung fase.
	unsigned long phase_coef = 1ul;
	unsigned long normal = (1ul << (width - tgt - 1));

	unsigned long ctl_bit = (1ul << tgt);
	for (int ctl=tgt+1; ctl<width; ctl++) {
		ctl_bit = (ctl_bit << 1);
		phase_coef = (phase_coef <<1);
		if ((i & ctl_bit) != 0) {
			phase_coef = phase_coef ^ 1ul;
		}
	}
	phase_coef = phase_coef ^ (normal);

	// Separuh warp kedua mendapat koefisien dari shared memory.
	if ((i%16)<8) {
		v_iother = sh_v[i_other-block_base];
		v_i = sh_v[i-block_base];
	}

	// Selesai menghitung gerbang.
	float phi = float(phase_coef) * float(M_PI) / float(normal);
	float c = cosf(phi);
	float s = sqrtf(1.0f-c*c);
	if (phi>float(M_PI)) {
		s *= -1.0f;
	}
	hipDoubleComplex phase = {c, s};
	v_i = hipCmul(v_i, phase);

	hipDoubleComplex ai, aother;	// koefisien i dan (i^tgt_bit)

	hipDoubleComplex cuM_SQRT1_2 = make_hipDoubleComplex(M_SQRT1_2, 0);
	ai = hipCmul(cuM_SQRT1_2, hipCsub(v_iother, v_i));
	aother = hipCmul(cuM_SQRT1_2, hipCadd(v_iother, v_i));



	if ((i%16)>=8) {
		sh_v[i-block_base] = ai;
		sh_v[i_other-block_base] = aother;
	} else {
		sh_v[i_other-block_base] = aother;
		sh_v[i-block_base] = ai;
	}
}


// Mengaplikasikan geser fase dan transformasi Hadamard untuk qubit 'tgt' dan state 'i'.
// Catatan: HANYA dipanggil dengan ((i & tgt_bit) == 1)
__device__ static void qft_gpu_v6_single_state(int tgt, unsigned long i, int width, hipDoubleComplex *v){
	unsigned long phase_coef = 1ul;
	unsigned long tgt_bit = (1ul << tgt);
	unsigned long i_other = i^tgt_bit;
	unsigned long normal = (1ul << (width - tgt - 1));

/*
	if ((i & tgt_bit) == 0) {
		// This function should not have been called in this case.
		return;
	}
*/

	// Catatan: Geser fase (dengan target sama) commute.
	unsigned long ctl_bit = (1ul << tgt);
	for (int ctl=tgt+1; ctl<width; ctl++) {
		ctl_bit = (ctl_bit << 1);
		phase_coef = (phase_coef <<1);
		if ((i & ctl_bit) != 0) {
			phase_coef = phase_coef ^ 1ul;
		}
	}
	phase_coef = phase_coef ^ (normal);
	float phi = float(phase_coef) * float(M_PI) / float(normal);
	float c = cosf(phi);
	float s = sqrtf(1.0f-c*c);
	if (phi>float(M_PI)) {
		s *= -1.0f;
	}
	hipDoubleComplex phase = {c, s};
	hipDoubleComplex v_i = v[i];
	hipDoubleComplex v_iother = v[i_other];
	v_i = hipCmul(v_i, phase);

	hipDoubleComplex ai, aother;	// koefisien i dan (i^tgt_bit)
	hipDoubleComplex cuM_SQRT1_2 = make_hipDoubleComplex(M_SQRT1_2, 0);
	ai = hipCmul(cuM_SQRT1_2, hipCsub(v_iother, v_i));
	aother = hipCmul(cuM_SQRT1_2, hipCadd(v_iother, v_i));

	v[i] = ai;
	v[i_other] = aother;
}


// Kernel ini melakukan QFT single stage.
__global__ static void K_qft_gpu_v6_stage(int width, hipDoubleComplex *v, int tgt){
	unsigned long N = (1ul << width);

	// Membagi threads ke tiap state.
	unsigned long long bidx = blockIdx.y*gridDim.x + blockIdx.x;
	unsigned long long i = bidx*blockDim.x + threadIdx.x;

	if (i >= N) {
		return;
	}

	unsigned long tgt_bit = (1ul << tgt);
	if ((i & tgt_bit) != 0) {
		qft_gpu_v6_single_state(tgt, i, width, v);
	}
}




// Kernel ini melakukan QFT single stage.
__global__ static void K_qft_gpu_v6_stage_0to8(int width, hipDoubleComplex *v){
	unsigned long N = (1ul << width);

	// Membagi threads ke tiap state.
	unsigned long long bidx = blockIdx.y*gridDim.x + blockIdx.x;
	unsigned long long i = bidx*blockDim.x + threadIdx.x;

	// salin ke dalam
	unsigned long block_base = bidx*blockDim.x;
	__shared__ hipDoubleComplex sh_v[512]; // Note: hardcoded shared memory size
	if (i < N) {
		sh_v[threadIdx.x] = v[i];
	}
	__syncthreads();


	// hitung
	for (int tgt = min(width-1,8); tgt >= 0; tgt--) {
		unsigned long tgt_bit = (1ul << tgt);
		if ( (i < N) && ((i & tgt_bit) != 0) ) {
			qft_gpu_v6_single_lower_state(tgt, i, width, block_base, sh_v);
		}
		__syncthreads();
	}

	// salin keluar
	if (i < N) {
		v[i] = sh_v[threadIdx.x];
	}
}

// Ini adalah bagian dari v3, yang akan kita gunakan untuk perhitungan gerbang.
__device__ static void qft_gpu_v3_single_state(int tgt, unsigned long i, int width,  hipDoubleComplex *v){
	unsigned long phase_coef = 1ul;
	unsigned long tgt_bit = (1ul << tgt);
	unsigned long i_other = i^tgt_bit;
	unsigned long normal = (1ul << (width - tgt - 1));

	if ((i & tgt_bit) == 0) {
		return;
	}

	// Catatan: Geser fase (dengan target sama) commute.
	unsigned long ctl_bit = (1ul << tgt);
	for (int ctl=tgt+1; ctl<width; ctl++) {
		ctl_bit = (ctl_bit << 1);
		phase_coef = (phase_coef <<1);
		if ((i & ctl_bit) != 0) {
			phase_coef = phase_coef ^ 1ul;
		}
	}
	phase_coef = phase_coef ^ (normal);
	float phi = float(phase_coef) * float(M_PI) / float(normal);

	float c = cosf(phi);
	float s = sqrtf(1.0f-c*c);
	if (phi>float(M_PI)) {
		s *= -1.0f;
	}
	hipDoubleComplex phase = {c, s};

	hipDoubleComplex v_i = v[i];
	hipDoubleComplex v_iother = v[i_other];
	v_i = hipCmul(v_i, phase);

	//hadamard_gpu(tgt, i, width, v);
	hipDoubleComplex ai, aother;	// coefficients i and (i^tgt_bit)

	hipDoubleComplex cuM_SQRT1_2 = make_hipDoubleComplex(M_SQRT1_2, 0);
	ai = hipCmul(cuM_SQRT1_2, hipCsub(v_iother, v_i));
	aother = hipCmul(cuM_SQRT1_2, hipCadd(v_iother, v_i));

	v[i] = ai;
	v[i_other] = aother;
}



__global__ static void K_qft_gpu_v6_stage_M_bits(int width, hipDoubleComplex *v, int tgt, int M){
	unsigned long N = (1ul << width);

	// Membagi threads ke tiap state.
	unsigned long long bidx = blockIdx.y*gridDim.x + blockIdx.x;
	unsigned long long i = bidx*blockDim.x + threadIdx.x;

	if (i >= N) {
		return;
	}

	// hitung
	int m;
	unsigned long mask = 0;
	for (m = 0; m < M; m++) {
		mask = mask | (1ul << (tgt-m));
	}

	unsigned long B = (1ul<<(tgt-(M-1)));
	int G = (1ul<<M);
	int tgtBit = tgt;
	if ( (i & mask) == mask) {
		for (m = (1ul<<(M-1)); m > 0; m = m>>1) {
			for (int g = G-1; g >= 0; g--) {
				if ((g & m) != 0) {
					unsigned long iother = i - B*(G-1-g);
					qft_gpu_v3_single_state(tgtBit, iother, width, v);
				}
			}
			tgtBit--;
		}
	}
}







// Implementasi QFT gerbang demi gerbang menggunakan GPU.
void qft_gpu_v6_helper(int width, hipDoubleComplex *d_v, int threadsPerBlock){
	// M adalah jumlah qubits yang diproses secara serentak;
	// Parameter ini bisa diubah
	int M=2;

	unsigned long N = (1ul << width);

	unsigned long long nblocks = (N+threadsPerBlock-1)/threadsPerBlock;
	unsigned long long xblocks, yblocks;

	yblocks = (nblocks+BLOCKS-1)/BLOCKS;
	xblocks = BLOCKS;
	if (nblocks < xblocks) {
		xblocks = nblocks;
	}
	dim3 blocks(xblocks, yblocks);

	// For each qubit...
	int band = 0;
	int tgt;
	int leftover = (width-9-band)%M;
	int uneven_stop = width - leftover;

	for (tgt=width-1; tgt>=uneven_stop; tgt--) {
		printf("tgt=%d\n", tgt);
		K_qft_gpu_v6_stage<<<blocks, threadsPerBlock>>>(width, d_v, tgt);
		CUT_CHECK_ERROR("K_qft_gpu_v6_stage failed.");
	}

	for (; tgt>=9+band; tgt-=M) {
		printf("stage_M_bits tgt=%d\n", tgt);
		K_qft_gpu_v6_stage_M_bits<<<blocks, threadsPerBlock>>>(width, d_v, tgt, M);
		CUT_CHECK_ERROR("K_qft_gpu_v6_stage_M_bits failed.");
	}

	for (; tgt>=9; tgt--) {
		printf("band: %d\n", tgt);
		K_qft_gpu_v6_stage<<<blocks, threadsPerBlock>>>(width, d_v, tgt);
		CUT_CHECK_ERROR("K_qft_gpu_v6_stage failed.");
	}

	K_qft_gpu_v6_stage_0to8<<<blocks, threadsPerBlock>>>(width, d_v);
	CUT_CHECK_ERROR("K_qft_gpu_v6_stage_0to8 failed.");
}
